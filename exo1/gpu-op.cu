#include "hip/hip_runtime.h"
/*********************************************************************************/
/* Matrix product program for a multi-core CPU and for a many-core GPU           */
/* S. Vialle - November 2016                                                     */
/*********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>

#include "main.h"
#include "gpu-op.h"


/*-------------------------------------------------------------------------------*/
/* GPU symbols                                                                   */
/*-------------------------------------------------------------------------------*/
__device__ T_real GPU_A[SIZE][SIZE];
__device__ T_real GPU_B[SIZE][SIZE];
__device__ T_real GPU_C[SIZE][SIZE];


/*-------------------------------------------------------------------------------*/
/* Init and finalize the GPU device.                                             */
/*-------------------------------------------------------------------------------*/
void gpuInit(void)
{
  hipInit(0);
}


void gpuFinalize(void)
{

}


/*-------------------------------------------------------------------------------*/
/* Transfer of CPU input data into GPU symbols                                   */
/*-------------------------------------------------------------------------------*/
void gpuSetDataOnGPU(void)
{
 // Set GPU_A symbol
 //CHECK_CUDA_SUCCESS(hipMemcpyToSymbol(...),
 //                   "Transfer A-->GPU_A");

 // Set GPU_B symbol
 // ...
}


/*-------------------------------------------------------------------------------*/
/* Transfer of GPU results into CPU array                                        */
/*-------------------------------------------------------------------------------*/
void gpuGetResultOnCPU(void)
{
 // Get GPU_C symbol
 // ...
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU.                                        */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v0(void)
{
 // Index computations
 //int lig = 
 //int col = 
 //T_real res = 0.0;

 // Matrix product computation
 // ...
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU.                                        */
/*-------------------------------------------------------------------------------*/
void gpuProduct(gkid_t kid)
{
 dim3 Dg, Db;

 switch(kid) {

 case GK0 : // Kernel v0 - using only global memory (with coalescent data accesses)
   // - init the grid of blocs
   //Db.x = ;
   //Db.y = ;
   //Db.z = ;
   //Dg.x = ;
   //Dg.y = ;
   //Dg.z = ;
   // - run the Grid of Blocs of threads
   //MatrixProductKernel_v0<<<Dg,Db>>>();
   break;

 case GK1 :
  break;

 case GK2 :
  break;
  
 case GK3 :
  break;

 case GK4 :
  break;
  
 case GK5 :
  break;

 default :
   fprintf(stderr,"Unknown GPU kernel!");
   exit(EXIT_FAILURE);
 }
}




